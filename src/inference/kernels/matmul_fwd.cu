#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

// Naive matrix multiplication kernel
__global__ void matmul_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Wrapper function for PyTorch
torch::Tensor matmul_forward(torch::Tensor A, torch::Tensor B) {
    const at::cuda::CUDAGuard device_guard(A.device());

    TORCH_CHECK(A.device().type() == torch::kCUDA, "A must be a CUDA tensor");
    TORCH_CHECK(B.device().type() == torch::kCUDA, "B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "B must be float32");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, "Both tensors must be 2D");
    TORCH_CHECK(A.size(1) == B.size(0), "Matrix dimensions don't match");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    auto C = torch::zeros({M, N}, torch::dtype(torch::kFloat32).device(A.device()));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matmul_kernel<<<numBlocks, threadsPerBlock>>>(
        A.data_ptr<float>(),
        B.data_ptr<float>(),
        C.data_ptr<float>(),
        M, N, K
    );

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}
